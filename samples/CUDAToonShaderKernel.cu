#include "hip/hip_runtime.h"

/***********************************************************************************/
/** INCLUSIONS                                                                    **/
/***********************************************************************************/

#include "../kernels/CUDAShaderCommon.cuh"

#include "CUDAToonShader.h"

/***********************************************************************************/
/** NOYAU                                                                         **/
/***********************************************************************************/

__global__ static void kernel_shade( uint* pixels, int pitch, int width, int height, float4 back_color, float4 cam_dir )
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

	if ( x >= width || y >= height ) return;

	int k = y * width + x;
	int n = y * pitch + x;

	float depth = tex1Dfetch( tex_depths, k );

	if ( depth < +HIP_NORM_HUGE_F )
	{
		float4 point = tex1Dfetch( tex_points, k );
		float4 normal = tex1Dfetch( tex_normals, k );

		cam_dir.w = 0.0f;
		normal.w = 0.0f;

		float toon_factor = fabs( 2.667f * dot( cam_dir, normal ) );

		if ( toon_factor < 0.6f ) toon_factor = 0.0f;
		else if ( toon_factor < 0.8f ) toon_factor = 0.2f;
		else if ( toon_factor < 1.0f ) toon_factor = 0.4f;
		else if ( toon_factor < 1.2f ) toon_factor = 0.6f;
		else if ( toon_factor < 1.4f ) toon_factor = 0.8f;
		else toon_factor = 1.0f;

		float4 point_color = make_float4( toon_factor );

		pixels[n] = make_pixel( point_color );
	}
	else
	{
		pixels[n] = make_pixel( back_color );
	}
}

/***********************************************************************************/
/** FONCTION                                                                      **/
/***********************************************************************************/

void runCUDAToonShader( CUDARenderSurface<uint>* output, CUDABuffer* input, CUDACamera* camera, CUDAToonShader* shader )
{
	dim3 db, dg;

	beginCUDAShader( output, input, db, dg );

	int pitch = output->getPitch() / sizeof(uint);
	int width = output->getWidth();
	int height = output->getHeight();

	uint* pixels = output->getPointer();

	float4 back_color = shader->getBackColor();

	float4 cam_dir = make_float4( camera->getDir() );

    kernel_shade<<<dg,db>>>
	(
		pixels,
		pitch,
		width,
		height,
		back_color,
		cam_dir
	);
	
	endCUDAShader();
}
